#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <cstdlib>
#include <time.h>

int main(void) {
    clock_t start, end;
    clock_t start_gen, end_gen;
    double total_time, total_time_gen;

    start_gen = clock();
    // generate 4GB of 32-bit random data
    thrust::host_vector<int> h_vec(16ul << 26);
    thrust::generate(h_vec.begin(), h_vec.end(), rand);
    end_gen = clock();
    printf("HELLO\n");

    start = clock();
    for (int i = 0; i<10; i++) {
    	//transfer data to the device
	thrust::device_vector<int> d_vec = h_vec;

	// sort data on the device
	thrust::sort(d_vec.begin(), d_vec.end());

	// transfer data back to host
	thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
    }			    
    end = clock();
    total_time = (end-start)/(double)CLOCKS_PER_SEC/10.0;
    total_time_gen = (end_gen-start_gen)/(double)CLOCKS_PER_SEC;
    printf("\nAverage time to sort 4GB of 32-bit integers: %fsec\n", total_time);
    printf("Total time to generate 4GB of 32-bit integers: %fsec\n", total_time_gen);    
    return 0;
}